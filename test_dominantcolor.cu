
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <iomanip>

using namespace std;

__global__ void add(int *a, int *b, int *c, int n){
	int index = threadIdx.x + blockIdx.x*blockDim.x;

	c[index] = a[index] + b[index];
}

__global__ void print(int *a){
	if( a[blockIdx.x] != 0)
 		printf("%d \n", blockIdx.x);
}

#define N (1024)
#define M (1000000)

int main(void){

	time_t timer = time(0);	

	int *a,*b,*c;					// host copies of a,b,c
	int *d_a, *d_b, *d_c;		// device copies of a,b,c
	int size = N * sizeof(int);

	// Allocate space for device copies of a,b,c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a,b,c and setup input

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for(int i=0; i<N; ++i)
	{
		a[i] = i*i;
		b[i] = i*2;
	}

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);  // Args: Dir. destino, Dir. origen, tamano de dato, sentido del envio
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<(N+M-1)/M,M>>> (d_a, d_b, d_c, N);
	print<<<N,1>>> (d_a);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
/*
	for(int i=0; i<N; ++i)
		std::cout << setw(6) << a[i];

	std::cout << std::endl;

	for(int i=0; i<N; ++i)
		std::cout << setw(6) << b[i];

	std::cout << std::endl;

	for(int i=0; i<N; ++i)
		std::cout << setw(6) << c[i];

	std::cout << std::endl;
*/
	// Cleanup
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	time_t timer2 = time(0);
	cout <<"Tiempo total: " << difftime(timer2, timer) << endl;

	return 0;
}